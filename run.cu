#include "hip/hip_runtime.h"
#include "run.h"

#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

extern double dt;
extern int N;

#define UPDATE_BODY(body, a_x, a_y) \
    body.vx += a_x * dt; \
    body.vy += a_y * dt; \
    body.x += body.vx * dt; \
    body.y += body.vy * dt;

// pthread worker
void *thread_worker(void *args)
{
    TaskParam *param = (TaskParam*)args;
    for (int i = param->start; i < param->end; i++) {
        double a_x = 0;
        double a_y = 0;
        param->tree->search(0, param->bodies[i], a_x, a_y);
        UPDATE_BODY(param->bodies[i], a_x, a_y);
    }

    pthread_exit(NULL);
}

// pthread version
void run_pthread_version(int i, int num_threads, Body *bodies,
    float *elapsed_time, QuadTree *tree)
{
    pthread_t *threads = (pthread_t*)malloc(sizeof(pthread_t) * num_threads);
    TaskParam *param = (TaskParam*)malloc(sizeof(TaskParam) * num_threads);
    int width = ceil((float)N / num_threads);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // build tree
    tree->build(bodies);

    // create threads to compute
    for (int j = 0; j < num_threads; j++) {
        param[j].start = j * width;
        if (j == num_threads - 1) {
            param[j].end = N;
        } else {
            param[j].end = (j + 1) * width;
        }
        param[j].bodies = bodies;
        param[j].tree = tree;
        pthread_create(threads + j, NULL, thread_worker, param + j);
    }

    // wait
    for (int j = 0; j < num_threads; j++) {
        pthread_join(threads[j], NULL);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(threads);
    free(param);
}
