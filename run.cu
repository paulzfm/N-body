#include "hip/hip_runtime.h"
#include "run.h"

#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

extern int N;
extern int n;
extern double threshold;

// pthread worker
void *thread_worker(void *args)
{
/*
    TaskParam *param = (TaskParam*)args;
    for (int i = param->start; i < param->end; i++) {
        param->tree->update(param->bodies + i);
    }

    pthread_exit(NULL);
*/
    pthread_exit(NULL);
}

// pthread version
void run_pthread_version(int i, int num_threads, Body *bodies,
    float *elapsed_time, Node *tree)
{
/*
    pthread_t *threads = (pthread_t*)malloc(sizeof(pthread_t) * num_threads);
    TaskParam *param = (TaskParam*)malloc(sizeof(TaskParam) * num_threads);
    int width = ceil((float)N / num_threads);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // build tree
    tree->build(bodies);

    // create threads to compute
    for (int j = 0; j < num_threads; j++) {
        param[j].start = j * width;
        if (j == num_threads - 1) {
            param[j].end = N;
        } else {
            param[j].end = (j + 1) * width;
        }
        param[j].bodies = bodies;
        param[j].tree = tree;
        pthread_create(threads + j, NULL, thread_worker, param + j);
    }

    // wait
    for (int j = 0; j < num_threads; j++) {
        pthread_join(threads[j], NULL);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(threads);
    free(param);
*/
}

// cuda worker
__global__ void cuda_worker(Node *tree, Body *bodies, double threshold, double size, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N) {
        return;
    }
    tree_update(bodies + i, tree, size, threshold);
}

// cuda version
void run_cuda_version(int i, Body *bodies,
    float *elapsed_time, Node *tree)
{
    hipEvent_t start, stop;
    Body *d_bodies;
    Node *d_tree;
    hipMalloc((void**)&d_bodies, sizeof(Body) * N);
    hipMalloc((void**)&d_tree, sizeof(Node) * n);
    double size;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // build tree
    tree_build(bodies, tree, N, &size);

    hipMemcpy(d_bodies, bodies, sizeof(Body) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_tree, tree, sizeof(Node) * n, hipMemcpyHostToDevice);

    // compute
    int block = ceil(N / 512.0);
    cuda_worker<<<block, 512>>>(d_tree, d_bodies, threshold, size, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(bodies, d_bodies, sizeof(Body) * N, hipMemcpyDeviceToHost);
}
