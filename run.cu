#include "hip/hip_runtime.h"
#include "run.h"

#include <pthread.h>
#include <stdlib.h>
#include <math.h>

extern "C" {
    void update_body(int i, Body *bodies, Body *new_body);
}

// pthread worker
void *thread_worker(void *args)
{
    TaskParam *param = (TaskParam*)args;
    int i;
    for (i = param->start; i < param->end; i++) {
        update_body(i, param->bodies, param->new_bodies + i);
    }

    pthread_exit(NULL);
}

// pthread version
void run_pthread_version(int i, int num_threads, Body *bodies,
    Body *new_bodies, float *elapsed_time)
{
    pthread_t *threads = (pthread_t*)malloc(sizeof(pthread_t) * num_threads);
    TaskParam *param = (TaskParam*)malloc(sizeof(TaskParam) * num_threads);
    int width = ceil((float)global.N / num_threads);
    int j;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // create threads to compute
    for (j = 0; j < num_threads; j++) {
        param[j].start = j * width;
        if (j == num_threads - 1) {
            param[j].end = global.N;
        } else {
            param[j].end = (j + 1) * width;
        }
        param[j].bodies = bodies;
        param[j].new_bodies = new_bodies;
        pthread_create(threads + j, NULL, thread_worker, param + j);
    }

    // wait
    for (j = 0; j < num_threads; j++) {
        pthread_join(threads[j], NULL);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsed_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(threads);
    free(param);
}
